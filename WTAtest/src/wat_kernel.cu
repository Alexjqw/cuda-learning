#include "hip/hip_runtime.h"
#include"../include/wta_kernel.hpp"
#include<iostream>

namespace srs{

    static constexpr unsigned int WARPS_PER_BLOCK = 8u;
    //static constexpr unsigned int WARP_SIZE = 32u;
    static constexpr unsigned int BLOCK_SIZE = WARPS_PER_BLOCK * WARP_SIZE;


    __device__ inline uint32_t pack_cost_index(uint32_t cost, uint32_t index) {
        union {
            uint32_t uint32;
            ushort2 uint16x2;
        } u;
        u.uint16x2.x = static_cast<uint16_t>(index);
        u.uint16x2.y = static_cast<uint16_t>(cost);
        return u.uint32;
    }

    __device__ uint32_t unpack_cost(uint32_t packed) {
        return packed >> 16;
    }

    __device__ int unpack_index(uint32_t packed) {
        return packed & 0xffffu;
    }

    using ComputeDisparity = uint32_t(*)(uint32_t, uint32_t, uint8_t*);

    __device__ inline uint32_t compute_disparity_normal(uint32_t disp, uint32_t cost = 0, uint8_t* smem = nullptr)
    {
        return disp;
    }

    template <size_t MAX_DISPARITY>
    __device__ inline uint32_t compute_disparity_subpixel(uint32_t disp, uint32_t cost, uint8_t* smem)
    {
        int subp = disp;
        subp <<= 4;
        if (disp > 0 && disp < MAX_DISPARITY - 1) {
            const int left = smem[disp - 1];
            const int right = smem[disp + 1];
            const int numer = left - right;
            const int denom = left - 2 * cost + right;
            subp += ((numer << 4) + denom) / (2 * denom);
        }
        return subp;
    }



    template <unsigned int MAX_DISPARITY, ComputeDisparity compute_disparity = compute_disparity_normal>
    __global__ void winner_takes_all_kernel(
        output_type *left_dest,
        const census_type *src,
        int width,
        int height,
        int pitch,
        float uniqueness)
    {
        static const unsigned int ACCUMULATION_PER_THREAD = 16u;
        static const unsigned int REDUCTION_PER_THREAD = MAX_DISPARITY / WARP_SIZE;
        static const unsigned int ACCUMULATION_INTERVAL = ACCUMULATION_PER_THREAD / REDUCTION_PER_THREAD;
        static const unsigned int UNROLL_DEPTH =
            (REDUCTION_PER_THREAD > ACCUMULATION_INTERVAL)
            ? REDUCTION_PER_THREAD
            : ACCUMULATION_INTERVAL;


        const unsigned int warp_id = threadIdx.x / WARP_SIZE;
        const unsigned int lane_id = threadIdx.x % WARP_SIZE;

        const unsigned int y = blockIdx.x * WARPS_PER_BLOCK + warp_id;
        src += y * MAX_DISPARITY * width;
        left_dest += y * pitch;


        if (y >= height) {
            return;
        }
        __shared__ uint8_t smem_cost_sum[WARPS_PER_BLOCK][ACCUMULATION_INTERVAL][MAX_DISPARITY];
        for (unsigned int x0 = 0; x0 < width; x0 += UNROLL_DEPTH) {
#pragma unroll
            for (unsigned int x1 = 0; x1 < UNROLL_DEPTH; ++x1) {
                if (x1 % ACCUMULATION_INTERVAL == 0) {
                    const unsigned int k = lane_id * ACCUMULATION_PER_THREAD;
                    const unsigned int k_hi = k / MAX_DISPARITY;
                    const unsigned int k_lo = k % MAX_DISPARITY;
                    const unsigned int x = x0 + x1 + k_hi;
                    if (x < width) {
                        const unsigned int offset = x * MAX_DISPARITY + k_lo;
                      /*  copy_data<ACCUMULATION_PER_THREAD>(
                            &smem_cost_sum[warp_id][k_hi][k_lo], &src[offset]);*/
                        uint32_t load_buffer[ACCUMULATION_PER_THREAD];
                        load_uint8_vector<ACCUMULATION_PER_THREAD>(
                            load_buffer, &src[offset]);
                        store_uint8_vector<ACCUMULATION_PER_THREAD>(
                            &smem_cost_sum[warp_id][k_hi][k_lo], load_buffer);
                    }
#if CUDA_VERSION >= 9000
                    __syncwarp();
#else
                    __threadfence_block();
#endif
                }
                const unsigned int x = x0 + x1;
                if (x < width) {
                    // Load sum of costs
                    const unsigned int smem_x = x1 % ACCUMULATION_INTERVAL;
                    const unsigned int k0 = lane_id * REDUCTION_PER_THREAD;
                    uint32_t local_cost_sum[REDUCTION_PER_THREAD];
                    load_uint8_vector<REDUCTION_PER_THREAD>(
                        local_cost_sum, &smem_cost_sum[warp_id][smem_x][k0]);
                    // Pack sum of costs and dispairty
                    uint32_t local_packed_cost[REDUCTION_PER_THREAD];
                    for (unsigned int i = 0; i < REDUCTION_PER_THREAD; ++i) {
                        local_packed_cost[i] = pack_cost_index(local_cost_sum[i], k0 + i);
                    }
                    // Update left
                    uint32_t best = 0xffffffffu;
                    for (unsigned int i = 0; i < REDUCTION_PER_THREAD; ++i) {
                        best = min(best, local_packed_cost[i]);
                    }
                    best = subgroup_min<WARP_SIZE>(best, 0xffffffffu);

#pragma unroll
                    // Resume updating left to avoid execution dependency
                    const uint32_t bestCost = unpack_cost(best);
                    const int bestDisp = unpack_index(best);
                    bool uniq = true;
                    for (unsigned int i = 0; i < REDUCTION_PER_THREAD; ++i) {
                        const uint32_t x = local_packed_cost[i];
                        const bool uniq1 = unpack_cost(x) * uniqueness >= bestCost;
                        const bool uniq2 = abs(unpack_index(x) - bestDisp) <= 1;
                        uniq &= uniq1 || uniq2;
                    }
                    uniq = subgroup_and<WARP_SIZE>(uniq, 0xffffffffu);
                    if (lane_id == 0) {
                        left_dest[x] = uniq ? compute_disparity(bestDisp, bestCost, smem_cost_sum[warp_id][smem_x]) : INVALID_DISP;
                    }
                }
            }
        }

    }


    void enqueue_winner_takes_all(
        output_type *left_dest,
        const census_type *src,
        int width,
        int height,
        int pitch,        
        float uniqueness,
        bool subpixel,
        hipStream_t stream)
    {
        const int gdim =
            (height + WARPS_PER_BLOCK - 1) / WARPS_PER_BLOCK;
        const int bdim = BLOCK_SIZE;
        if (subpixel) {
            winner_takes_all_kernel<64, compute_disparity_subpixel<64>> << <gdim, bdim, 0, stream >> > (
                left_dest, src, width, height, pitch, uniqueness);
        }
        else
        {
            winner_takes_all_kernel<64, compute_disparity_normal> << <gdim, bdim, 0, stream >> > (
                left_dest, src, width, height, pitch, uniqueness);

        }
    }

}

