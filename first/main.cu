#include"include/kener.hpp"
#include "include/book.hpp"


int main(void) {
    int a[N], b[N], c[N];
    int* dev_a, * dev_b, * dev_c;

    // allocate the memory on the GPU
    CHECK(hipMalloc((void**)&dev_a, N * sizeof(int)));
    CHECK(hipMalloc((void**)&dev_b, N * sizeof(int)));
    CHECK(hipMalloc((void**)&dev_c, N * sizeof(int)));

    // fill the arrays 'a' and 'b' on the CPU
    for (int i = 0; i < N; i++) {
        a[i] = -i;
        b[i] = i * i;
    }

    // copy the arrays 'a' and 'b' to the GPU
    CHECK(hipMemcpy(dev_a, a, N * sizeof(int),
        hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dev_b, b, N * sizeof(int),
        hipMemcpyHostToDevice));

    add << <N, 1 >> > (dev_a, dev_b, dev_c);

    // copy the array 'c' back from the GPU to the CPU
    CHECK(hipMemcpy(c, dev_c, N * sizeof(int),
        hipMemcpyDeviceToHost));

    // display the results
    for (int i = 0; i < N; i++) {
        printf("%d + %d = %d\n", a[i], b[i], c[i]);
    }

    // free the memory allocated on the GPU
    CHECK(hipFree(dev_a));
    CHECK(hipFree(dev_b));
    CHECK(hipFree(dev_c));

    return 0;
}

