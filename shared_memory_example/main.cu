#include"include/kernel.hpp"
#include "include/common.hpp"

int main(void)
{
    real* h_x = (real*)malloc(M);
    for (int n = 0; n < N; ++n)
    {
        h_x[n] = 1.23;
    }
    real* d_x;
    CHECK(hipMalloc(&d_x, M));

    printf("\nUsing global memory only:\n");
    timing(h_x, d_x, 0);
    printf("\nUsing static shared memory:\n");
    timing(h_x, d_x, 1);
    printf("\nUsing dynamic shared memory:\n");
    timing(h_x, d_x, 2);

    free(h_x);
    CHECK(hipFree(d_x));
    return 0;
}