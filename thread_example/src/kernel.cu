#include "hip/hip_runtime.h"
#include"../include/kernel.hpp"
#include<stdio.h>

void timing(const real* d_A, real* d_B, const int N, const int task)
{
    const int grid_size_x = (N + TILE_DIM - 1) / TILE_DIM;
    const int grid_size_y = grid_size_x;
    const dim3 block_size(TILE_DIM, TILE_DIM);
    const dim3 grid_size(grid_size_x, grid_size_y);

    float t_sum = 0;
    float t2_sum = 0;
    for (int repeat = 0; repeat <= NUM_REPEATS; ++repeat)
    {
        hipEvent_t start, stop;
        CHECK(hipEventCreate(&start));
        CHECK(hipEventCreate(&stop));
        CHECK(hipEventRecord(start));
        hipEventQuery(start);

        switch (task)
        {
        case 0:
            copy << <grid_size, block_size >> > (d_A, d_B, N);
            break;
        case 1:
            transpose1 << <grid_size, block_size >> > (d_A, d_B, N);
            break;
        case 2:
            transpose2 << <grid_size, block_size >> > (d_A, d_B, N);
            break;
        case 3:
            transpose3 << <grid_size, block_size >> > (d_A, d_B, N);
            break;
        default:
            printf("Error: wrong task\n");
            exit(1);
            break;
        }

        CHECK(hipEventRecord(stop));
        CHECK(hipEventSynchronize(stop));
        float elapsed_time;
        CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
        printf("Time = %g ms.\n", elapsed_time);

        if (repeat > 0)
        {
            t_sum += elapsed_time;
            t2_sum += elapsed_time * elapsed_time;
        }

        CHECK(hipEventDestroy(start));
        CHECK(hipEventDestroy(stop));
    }

    const float t_ave = t_sum / NUM_REPEATS;
    const float t_err = sqrt(t2_sum / NUM_REPEATS - t_ave * t_ave);
    printf("Time = %g +- %g ms.\n", t_ave, t_err);
}

__global__ void copy(const real* A, real* B, const int N)
{
    const int nx = blockIdx.x * TILE_DIM + threadIdx.x;
    const int ny = blockIdx.y * TILE_DIM + threadIdx.y;
    const int index = ny * N + nx;
    if (nx < N && ny < N)
    {
        B[index] = A[index];
    }
}

__global__ void transpose1(const real* A, real* B, const int N)
{
    const int nx = blockIdx.x * blockDim.x + threadIdx.x;
    const int ny = blockIdx.y * blockDim.y + threadIdx.y;
    if (nx < N && ny < N)
    {
        B[nx * N + ny] = A[ny * N + nx];
    }
}

__global__ void transpose2(const real* A, real* B, const int N)
{
    const int nx = blockIdx.x * blockDim.x + threadIdx.x;
    const int ny = blockIdx.y * blockDim.y + threadIdx.y;
    if (nx < N && ny < N)
    {
        B[ny * N + nx] = A[nx * N + ny];
    }
}

__global__ void transpose3(const real* A, real* B, const int N)
{
    const int nx = blockIdx.x * blockDim.x + threadIdx.x;
    const int ny = blockIdx.y * blockDim.y + threadIdx.y;
    if (nx < N && ny < N)
    {
        B[ny * N + nx] = __ldg(&A[nx * N + ny]);
    }
}

void print_matrix(const int N, const real* A)
{
    for (int ny = 0; ny < N; ny++)
    {
        for (int nx = 0; nx < N; nx++)
        {
            printf("%g\t", A[ny * N + nx]);
        }
        printf("\n");
    }
}

