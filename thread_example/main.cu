#include "hip/hip_runtime.h"
#include"include/kernel.hpp"
#include "include/common.hpp"




int main(int argc, char** argv)
{
    if (argc != 2)
    {
        printf("usage: %s N\n", argv[0]);
        exit(1);
    }
    const int N = atoi(argv[1]);

    const int N2 = N * N;
    const int M = sizeof(real) * N2;
    real* h_A = (real*)malloc(M);
    real* h_B = (real*)malloc(M);
    for (int n = 0; n < N2; ++n)
    {
        h_A[n] = n;
    }
    real* d_A, * d_B;
    CHECK(hipMalloc(&d_A, M));
    CHECK(hipMalloc(&d_B, M));
    CHECK(hipMemcpy(d_A, h_A, M, hipMemcpyHostToDevice));

    printf("\ncopy:\n");
    timing(d_A, d_B, N, 0);
    printf("\ntranspose with coalesced read:\n");
    timing(d_A, d_B, N, 1);
    printf("\ntranspose with coalesced write:\n");
    timing(d_A, d_B, N, 2);
    printf("\ntranspose with coalesced write and __ldg read:\n");
    timing(d_A, d_B, N, 3);

    CHECK(hipMemcpy(h_B, d_B, M, hipMemcpyDeviceToHost));
    if (N <= 10)
    {
        printf("A =\n");
        print_matrix(N, h_A);
        printf("\nB =\n");
        print_matrix(N, h_B);
    }

    free(h_A);
    free(h_B);
    CHECK(hipFree(d_A));
    CHECK(hipFree(d_B));
    return 0;
}

